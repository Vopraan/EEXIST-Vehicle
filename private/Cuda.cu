#include "hip/hip_runtime.h"
#include "../public/Cuda.h"

void StartSysUpdate(struct System* SystemPtr, struct tube*** BufferPtr)
{
	//printf("WHAT\n");

	UpdateSystems /*<<< SIZE, SIZE >>>*/ (SystemPtr, BufferPtr);
	//hipDeviceSynchronize();
}


// PERFROM DATA TRNASFERS FOR UPDATE
//__global__
void UpdateSystems(struct System* SystemPtr, struct tube*** BufferPtr)
{
	for(int y = 0; y < SIZE; y++)
	{
		for(int x = 0; x < SIZE; x++)
		{
			// STORE TAKE AND PUT CHEMC COORDS
			int TakeSrcX = ((int)SystemPtr->Tubes[x][y].SrcX + SystemPtr->Biases[x][y].SrcX) % SIZE;
			int TakeSrcY = ((int)SystemPtr->Tubes[x][y].SrcY + SystemPtr->Biases[x][y].SrcY) % SIZE;
			int TakeDesX = ((int)SystemPtr->Tubes[x][y].DesX + SystemPtr->Biases[x][y].DesX) % SIZE;
			int TakeDesY = ((int)SystemPtr->Tubes[x][y].DesY + SystemPtr->Biases[x][y].DesY) % SIZE;

			// LOOP THROUGH KARMA TUBES FOR CENTERAL TUBE TRANSFER
			for(int Ky = -KARMA; Ky <= KARMA; Ky++)
			{
				for (int Kx = -KARMA; Kx <= KARMA; Kx++)
				{
					float KarmDist = sqrtf(powf(Kx, 2) + pow(Ky, 2));
					if(KarmDist > KARMA) { continue; }

					// ADD AKRMA OFFSET TO CENTERAL TUBE
					int KSrcX = TakeSrcX + Kx;
					int KScrY = TakeSrcY + Ky;
					int KDesX = TakeDesX + Kx;
					int KDesY = TakeDesY + Ky;

					// FILTER OUT TRNASFERS FROM FIRST TWO ROWS THE FIRST ROW IS RESERVED AS INPUT FOR THE SYSTEM
					if(KScrY == 0 || KDesY == 0) { continue; }

					// CALC HOW MUCH CHEM TO TAKE
					float Take = TAKE * (1 - (KarmDist / KARMA));

					// MAKE SURE TANSFER IS IN RANGE OF TUBES
					KSrcX = abs(KSrcX % SIZE);
					KScrY = abs(KScrY % SIZE);
					KDesX = abs(KDesX % SIZE);
					KDesY = abs(KDesY % SIZE);

					if(SystemPtr->Tubes[KSrcX][KScrY].SrcX > Take && (SystemPtr->Tubes[KDesX][KDesY].SrcX + Take) < SIZE)
					{
						SystemPtr->Tubes[KSrcX][KScrY].SrcX = SystemPtr->Tubes[KSrcX][KScrY].SrcX - Take;
						SystemPtr->Tubes[KSrcX][KScrY].SrcY = SystemPtr->Tubes[KSrcX][KScrY].SrcY - Take;
						SystemPtr->Tubes[KSrcX][KScrY].DesX = SystemPtr->Tubes[KSrcX][KScrY].DesX - Take;
						SystemPtr->Tubes[KSrcX][KScrY].DesY = SystemPtr->Tubes[KSrcX][KScrY].DesY - Take;

						SystemPtr->Tubes[KDesX][KDesY].SrcX = SystemPtr->Tubes[KDesX][KDesY].SrcX + Take;
						SystemPtr->Tubes[KDesX][KDesY].SrcY = SystemPtr->Tubes[KDesX][KDesY].SrcY + Take;
						SystemPtr->Tubes[KDesX][KDesY].DesX = SystemPtr->Tubes[KDesX][KDesY].DesX + Take;
						SystemPtr->Tubes[KDesX][KDesY].DesY = SystemPtr->Tubes[KDesX][KDesY].DesY + Take;
					}
				}
			}
		}
	}


	/*
	int x = blockIdx.x;
	int y = threadIdx.x;

	// STORE TAKE AND PUT CHEMC COORDS
	int TakeSrcX = ((int)SystemPtr->Tubes[x][y].SrcX + SystemPtr->Biases[x][y].SrcX) % SIZE;
	int TakeSrcY = ((int)SystemPtr->Tubes[x][y].SrcY + SystemPtr->Biases[x][y].SrcY) % SIZE;
	int TakeDesX = ((int)SystemPtr->Tubes[x][y].DesX + SystemPtr->Biases[x][y].DesX) % SIZE;
	int TakeDesY = ((int)SystemPtr->Tubes[x][y].DesY + SystemPtr->Biases[x][y].DesY) % SIZE;

	// LOOP THROUGH KARMA TUBES FOR CENTERAL TUBE TRANSFER
	for(int Ky = -KARMA; Ky <= KARMA; Ky++)
	{
		for (int Kx = -KARMA; Kx <= KARMA; Kx++)
		{
			float KarmDist = sqrtf(powf(Kx, 2) + pow(Ky, 2));
			if(KarmDist > KARMA) { continue; }

			// ADD AKRMA OFFSET TO CENTERAL TUBE
			int KSrcX = TakeSrcX + Kx;
			int KScrY = TakeSrcY + Ky;
			int KDesX = TakeDesX + Kx;
			int KDesY = TakeDesY + Ky;

			// FILTER OUT TRNASFERS FROM FIRST TWO ROWS THE FIRST ROW IS RESERVED AS INPUT FOR THE SYSTEM
			if(KScrY == 0 || KDesY == 0) { continue; }

			// CALC HOW MUCH CHEM TO TAKE
			float Take = TAKE * (1 - (KarmDist / KARMA));

			// MAKE SURE TANSFER IS IN RANGE OF TUBES
			KSrcX = abs(KSrcX % SIZE);
			KScrY = abs(KScrY % SIZE);
			KDesX = abs(KDesX % SIZE);
			KDesY = abs(KDesY % SIZE);

			int i = (y * SIZE) + x;

			BufferPtr[i][KSrcX][KScrY].SrcX = -Take;
			BufferPtr[i][KSrcX][KScrY].SrcY = -Take;
			BufferPtr[i][KSrcX][KScrY].DesX = -Take;
			BufferPtr[i][KSrcX][KScrY].DesY = -Take;

			BufferPtr[i][KDesX][KDesY].SrcX =  Take;
			BufferPtr[i][KDesX][KDesY].SrcY =  Take;
			BufferPtr[i][KDesX][KDesY].DesX =  Take;
			BufferPtr[i][KDesX][KDesY].DesY =  Take;
		}
	}
	*/
}

void StartSysAdd(struct System* SystemPtr, struct tube*** BufferPtr)
{
	//AddSystems <<< SIZE, SIZE >>> (SystemPtr, BufferPtr);
	//hipDeviceSynchronize();
}

// ADD TOGETHER GIVEN MATRICIES
__global__
void AddSystems(struct System* SystemPtr, struct tube*** BufferPtr)
{
	int x = blockIdx.x;
	int y = threadIdx.x;

	if(y == 0) {return;}

	//Add MAtricies
	for(int i = 0; i < SIZE * SIZE; i++)
	{
		SystemPtr->Tubes[x][y].SrcX = SystemPtr->Tubes[x][y].SrcX + BufferPtr[i][x][y].SrcX;
		SystemPtr->Tubes[x][y].SrcY = SystemPtr->Tubes[x][y].SrcY + BufferPtr[i][x][y].SrcY;
		SystemPtr->Tubes[x][y].DesX = SystemPtr->Tubes[x][y].DesX + BufferPtr[i][x][y].DesX;
		SystemPtr->Tubes[x][y].DesY = SystemPtr->Tubes[x][y].DesY + BufferPtr[i][x][y].DesY;
	}

	//Calmp Tube VAlues
	if(SystemPtr->Tubes[x][y].SrcX < 0) 		{ SystemPtr->Tubes[x][y].SrcX = 0; }
	if(SystemPtr->Tubes[x][y].SrcX > SIZE - 1) 	{ SystemPtr->Tubes[x][y].SrcX = SIZE - 1; }

	if(SystemPtr->Tubes[x][y].SrcY < 0) 		{ SystemPtr->Tubes[x][y].SrcY = 0; }
	if(SystemPtr->Tubes[x][y].SrcY > SIZE - 1) 	{ SystemPtr->Tubes[x][y].SrcY = SIZE - 1; }

	if(SystemPtr->Tubes[x][y].DesX < 0) 		{ SystemPtr->Tubes[x][y].DesX = 0; }
	if(SystemPtr->Tubes[x][y].DesX > SIZE - 1) 	{ SystemPtr->Tubes[x][y].DesX = SIZE - 1; }

	if(SystemPtr->Tubes[x][y].DesY < 0) 		{ SystemPtr->Tubes[x][y].DesY = 0; }
	if(SystemPtr->Tubes[x][y].DesY > SIZE - 1) 	{ SystemPtr->Tubes[x][y].DesY = SIZE - 1; }
}